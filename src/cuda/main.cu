#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>

#include "utils.h"
#include "reader.h"
#include "landsat.h"
#include "constants.h"
#include "parameters.h"

/** 
 * @brief Main function
 * This function is responsible for reading the input parameters and calling the Landsat class to process the products.
 * 
 * @param argc Number of input parameters
 * @param argv Input parameters
 *              - INPUT_BAND_BLUE_INDEX    = 1;
 *              - INPUT_BAND_GREEN_INDEX   = 2;
 *              - INPUT_BAND_RED_INDEX     = 3;
 *              - INPUT_BAND_NIR_INDEX     = 4;
 *              - INPUT_BAND_SWIR1_INDEX   = 5;
 *              - INPUT_BAND_TERMAL_INDEX  = 6;
 *              - INPUT_BAND_SWIR2_INDEX   = 7;
 *              - INPUT_BAND_TAL_INDEX     = 8;
 *              - INPUT_MTL_DATA_INDEX     = 9;
 *              - INPUT_STATION_DATA_INDEX = 10;
 *              - INPUT_LAND_COVER_INDEX   = 11;
 *              - OUTPUT_FOLDER            = 12;
 * @return int
*/
int main(int argc, char *argv[])
{
  int INPUT_BAND_TAL_INDEX    = 8;
  int INPUT_MTL_DATA_INDEX     = 9;
  int INPUT_STATION_DATA_INDEX = 10;
  int INPUT_LAND_COVER_INDEX   = 11;
  int OUTPUT_FOLDER            = 12;
  int METHOD_INDEX             = 13;
  int THREADS_INDEX            = 14;
  int BLOCKS_INDEX             = 15;

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);

  string path_meta_file = argv[INPUT_MTL_DATA_INDEX];
  string station_data_path = argv[INPUT_STATION_DATA_INDEX];
  string land_cover_path = argv[INPUT_LAND_COVER_INDEX];

  // load bands path
  string bands_paths[INPUT_BAND_TAL_INDEX + 1];
  for (int i = 1; i <= INPUT_BAND_TAL_INDEX; i++) {
    bands_paths[i] = argv[i];
  }

  // load selected method 
  int method = 0;
  if(argc >= 14){
    string flag = argv[METHOD_INDEX];
    if(flag.substr(0, 6) == "-meth=")
      method = flag[6] - '0';
  }

  // load threads number
  int threads_num = 1;
  if(argc >= 15){
    string threads_flag = argv[THREADS_INDEX];
    if(threads_flag.substr(0,9) == "-threads=")
      threads_num = atof(threads_flag.substr(9, threads_flag.size()).c_str());
  }

  // load blocks number
  int blocks_num = deviceProp.maxBlocksPerMultiProcessor;
  if(argc >= 16){
    string blocks_flag = argv[BLOCKS_INDEX];
    if(blocks_flag.substr(0,8) == "-blocks=")
      blocks_num = atof(blocks_flag.substr(8, blocks_flag.size()).c_str());
  }

  // load output folder
  string output_folder = argv[OUTPUT_FOLDER];
  string output_time = output_folder + "/time.csv";  
  string output_metadata = output_folder + "/metadata.txt";
  string output_products = output_folder + "/products.txt";

  // =====  START + TIME OUTPUT =====
  MTL mtl = MTL(path_meta_file);
  Sensor sensor = Sensor(mtl.number_sensor, mtl.year);
  Station station = Station(station_data_path, mtl.image_hour);
  Landsat landsat = Landsat(bands_paths, land_cover_path, mtl);

  ofstream time_output;
  time_output.open(output_time);
  string general, initial, final;
  system_clock::time_point begin, end;
  int64_t initial_time, final_time, general_time;

  begin = system_clock::now();
  initial_time = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();

  time_output << "PHASE,TIMESTAMP,START_TIME,END_TIME" << std::endl;
  time_output << landsat.compute_Rn_G(sensor, station);
  time_output << landsat.select_endmembers(method);
  time_output << landsat.converge_rah_cycle(station, method, threads_num, blocks_num);
  time_output << landsat.compute_H_ET(station);

  end = system_clock::now();
  final_time = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
  general_time = duration_cast<milliseconds>(end.time_since_epoch() - begin.time_since_epoch()).count();
  time_output << "TOTAL," << general_time << "," << initial_time << "," << final_time << std::endl;
  time_output.close();

  // landsat.save_products(output_products);
  // landsat.print_products(output_products);
  // landsat.read_products(output_folder);
  landsat.close();

  // =====  END + METADATA OUTPUT =====
  ofstream metadata_output;
  metadata_output.open(output_metadata);
  metadata_output << "Image height: " << landsat.height_band << std::endl;
  metadata_output << "Image width: " << landsat.width_band << std::endl;
  metadata_output << "informed threads: " << threads_num << std::endl;
  metadata_output << "informed blocks: " << blocks_num << std::endl;
  metadata_output << "The GPU is a " << deviceProp.name << std::endl;
  metadata_output << "The GPU has " << deviceProp.multiProcessorCount << " SMs" << std::endl;
  metadata_output << "The GPU has " << deviceProp.persistingL2CacheMaxSize  << " bytes of L2 cache" << std::endl;
  metadata_output << "The GPU has " << deviceProp.concurrentKernels << " concurrent kernels" << std::endl;
  metadata_output << "The GPU has " << deviceProp.maxBlocksPerMultiProcessor << " max blocks per SM" << std::endl;
  metadata_output << "The GPU has " << deviceProp.maxThreadsPerMultiProcessor << " max threads per SM" << std::endl;
  metadata_output << "The GPU has " << deviceProp.maxGridSize << " max grid size" << std::endl;
  metadata_output.close();

  return 0;
}
