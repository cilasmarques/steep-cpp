#include "hip/hip_runtime.h"
#include "landsat.h"

Landsat::Landsat(string bands_paths[], string land_cover_path, MTL mtl)
{
  Reader TIFFs_reader = Reader();

  for (int i = 1; i <= 8; i++)
  {
    string path_tiff_base = bands_paths[i];
    this->bands_resampled[i] = TIFFOpen(path_tiff_base.c_str(), "rm");
    TIFFs_reader.check_open_tiff(this->bands_resampled[i]);
  }

  uint16_t sample_format;
  uint32_t height, width;
  TIFFGetField(bands_resampled[1], TIFFTAG_IMAGELENGTH, &height);
  TIFFGetField(bands_resampled[1], TIFFTAG_IMAGEWIDTH, &width);
  TIFFGetField(bands_resampled[1], TIFFTAG_SAMPLEFORMAT, &sample_format);

  this->width_band = width;
  this->height_band = height;
  this->sample_bands = sample_format;

  this->mtl = mtl;
  this->products = Products(this->width_band, this->height_band);
};

string Landsat::select_endmembers(int method)
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  if (method == 0)
  { // STEEP
	  pair<Candidate, Candidate> pixels = getEndmembersSTEPP(products.ndvi_vector, products.surface_temperature_vector, products.albedo_vector, products.net_radiation_vector, products.soil_heat_vector, height_band, width_band);
		hot_pixel = pixels.first; cold_pixel = pixels.second;
  }
  else if (method == 1)
  { // ASEBAL
	  pair<Candidate, Candidate> pixels = getEndmembersASEBAL(products.ndvi_vector, products.surface_temperature_vector, products.albedo_vector, products.net_radiation_vector, products.soil_heat_vector, height_band, width_band);
		hot_pixel = pixels.first; cold_pixel = pixels.second;
  }

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  return "P2 - PIXEL SELECTION," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
}

string Landsat::converge_rah_cycle(Station station, int method, int threads_per_block)
{
  string result = "";
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  double ustar_station = (VON_KARMAN * station.v6) / (log(station.WIND_SPEED / station.SURFACE_ROUGHNESS));
  double u10 = (ustar_station / VON_KARMAN) * log(10 / station.SURFACE_ROUGHNESS);
  double ndvi_min = 1.0;
  double ndvi_max = -1.0;

  for (int line = 0; line < this->height_band; line++)
  {
    vector<double> ndvi_line = products.ndvi_vector[line];
    for (int col = 0; col < this->width_band; col++)
    {
      if (ndvi_line[col] < ndvi_min)
        ndvi_min = ndvi_line[col];
      if (ndvi_line[col] > ndvi_max)
        ndvi_max = ndvi_line[col];
    }
  }

  for (int line = 0; line < height_band; line++)
  {
    products.d0_fuction(line);
    products.zom_fuction(station.A_ZOM, station.B_ZOM, line);
    products.ustar_fuction(u10, line);
    products.kb_function(ndvi_max, ndvi_min, line);
    products.aerodynamic_resistance_fuction(line);
  }

  ofstream outFile;

  outFile.open("./input/raw/ZOM.dat", std::ios::binary);
  outFile.write(reinterpret_cast<char*>(products.zom_pointer), height_band * width_band * sizeof(double));
  outFile.close();

  outFile.open("./input/raw/D0.dat", std::ios::binary);
  outFile.write(reinterpret_cast<char*>(products.d0_pointer), height_band * width_band * sizeof(double));
  outFile.close();

  outFile.open("./input/raw/KB1.dat", std::ios::binary);
  outFile.write(reinterpret_cast<char*>(products.kb1_pointer), height_band * width_band * sizeof(double));
  outFile.close();

  outFile.open("./input/raw/USTAR.dat", std::ios::binary);
  outFile.write(reinterpret_cast<char*>(products.ustar_pointer), height_band * width_band * sizeof(double));
  outFile.close();

  outFile.open("./input/raw/AERODYNAMIC_RESISTANCE.dat", std::ios::binary);
  outFile.write(reinterpret_cast<char*>(products.aerodynamic_resistance_pointer), height_band * width_band * sizeof(double));
  outFile.close();

  result += products.rah_correction_function_blocks(ndvi_min, ndvi_max, hot_pixel, cold_pixel, threads_per_block);

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  result += "P2 - RAH CYCLE," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
  return result;
};


string Landsat::compute_Rn_G(Sensor sensor, Station station)
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  TIFF *tal = this->bands_resampled[8];
  for (int line = 0; line < height_band; line++)
  {
    tdata_t tal_line_buff = _TIFFmalloc(TIFFScanlineSize(tal));
    unsigned short curr_tal_line_size = TIFFScanlineSize(tal) / width_band;
    Reader tal_reader = Reader(sample_bands, curr_tal_line_size, tal_line_buff);
    TIFFReadScanline(tal, tal_line_buff, line);

    products.radiance_function(bands_resampled, width_band, sample_bands, mtl, sensor, line);
    products.reflectance_function(bands_resampled, width_band, sample_bands, mtl, sensor, line);
    products.albedo_function(tal_reader, sensor, width_band, mtl.number_sensor, line);

    // Vegetation indices
    products.ndvi_function(width_band, line);
    products.pai_function(width_band, line);
    products.lai_function(width_band, line);
    products.evi_function(width_band, line);

    // Emissivity indices
    products.enb_emissivity_function(width_band, line);
    products.eo_emissivity_function(width_band, line);
    products.ea_emissivity_function(tal_reader, width_band, line);
    products.surface_temperature_function(mtl.number_sensor, width_band, line);

    // Radiation waves
    products.short_wave_radiation_function(tal_reader, mtl, width_band, line);
    products.large_wave_radiation_surface_function(width_band, line);
    products.large_wave_radiation_atmosphere_function(width_band, station.temperature_image, line);

    // Main products
    products.net_radiation_function(width_band, line);
    products.soil_heat_flux_function(width_band, line);

    _TIFFfree(tal_line_buff);
  }
  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  ofstream outFile;

  double ndvi_pointer[height_band][width_band];
  double albedo_pointer[height_band][width_band];
  double net_radiation_pointer[height_band][width_band];
  double soil_heat_pointer[height_band][width_band];
  double ts_pointer[height_band][width_band];

  for (int i = 0; i < height_band; i++)
  {
    for (int j = 0; j < width_band; j++)
    {
      ndvi_pointer[i][j] = products.ndvi_vector[i][j];
      albedo_pointer[i][j] = products.albedo_vector[i][j];
      net_radiation_pointer[i][j] = products.net_radiation_vector[i][j];
      soil_heat_pointer[i][j] = products.soil_heat_vector[i][j];
      ts_pointer[i][j] = products.surface_temperature_vector[i][j];
    }
  }

  outFile.open("./input/raw/NDVI.dat", std::ios::binary);
  outFile.write(reinterpret_cast<char*>(ndvi_pointer), height_band * width_band * sizeof(double));
  outFile.close();

  outFile.open("./input/raw/ALBEDO.dat", std::ios::binary);
  outFile.write(reinterpret_cast<char*>(albedo_pointer), height_band * width_band * sizeof(double));
  outFile.close();

  outFile.open("./input/raw/NET_RADIATION.dat", std::ios::binary);
  outFile.write(reinterpret_cast<char*>(net_radiation_pointer), height_band * width_band * sizeof(double));
  outFile.close();

  outFile.open("./input/raw/SOIL_HEAT_FLUX.dat", std::ios::binary);
  outFile.write(reinterpret_cast<char*>(soil_heat_pointer), height_band * width_band * sizeof(double));
  outFile.close();

  outFile.open("./input/raw/TS.dat", std::ios::binary);
  outFile.write(reinterpret_cast<char*>(ts_pointer), height_band * width_band * sizeof(double));
  outFile.close();

  return "P1 - Rn_G," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
}

string Landsat::compute_H_ET(Station station)
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  double dr = (1 / mtl.distance_earth_sun) * (1 / mtl.distance_earth_sun);
  double sigma = 0.409 * sin(((2 * PI / 365) * mtl.julian_day) - 1.39);
  double phi = (PI / 180) * station.latitude;
  double omegas = acos(-tan(phi) * tan(sigma));
  double Ra24h = (((24 * 60 / PI) * GSC * dr) * (omegas * sin(phi) * sin(sigma) + cos(phi) * cos(sigma) * sin(omegas))) * (1000000 / 86400.0);
  double Rs24h = station.INTERNALIZATION_FACTOR * sqrt(station.v7_max - station.v7_min) * Ra24h;

  double dt_pq_terra = products.H_pq_terra * products.rah_ini_pq_terra / (RHO * SPECIFIC_HEAT_AIR);
  double dt_pf_terra = products.H_pf_terra * products.rah_ini_pf_terra / (RHO * SPECIFIC_HEAT_AIR);

  double b = (dt_pq_terra - dt_pf_terra) / (hot_pixel.temperature - cold_pixel.temperature);
  double a = dt_pf_terra - (b * (cold_pixel.temperature - 273.15));

  for (int line = 0; line < height_band; line++)
  {
    products.sensible_heat_flux_function(a, b, line);
    products.latent_heat_flux_function(width_band, line);
    products.net_radiation_24h_function(Ra24h, Rs24h, width_band, line);
    products.evapotranspiration_fraction_fuction(width_band, line);
    products.sensible_heat_flux_24h_fuction(width_band, line);
    products.latent_heat_flux_24h_function(width_band, line);
    products.evapotranspiration_24h_function(station, width_band, line);
    products.evapotranspiration_function(width_band, line);
  }
  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  return "P2 - FINAL PRODUCTS," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
};

void Landsat::save_products(string output_path)
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  std::ofstream outputProds(output_path);
  std::streambuf *coutProds = std::cout.rdbuf();
  std::cout.rdbuf(outputProds.rdbuf());

  std::cout << " ==== albedo" << std::endl;
  printVector2x2(products.albedo_vector);

  std::cout << " ==== ndvi" << std::endl;
  printVector2x2(products.ndvi_vector);

  std::cout << " ==== net_radiation" << std::endl;
  printVector2x2(products.net_radiation_vector);

  std::cout << " ==== soil_heat" << std::endl;
  printVector2x2(products.soil_heat_vector);

  std::cout << " ==== sensible_heat_flux" << std::endl;
  printVector2x2(products.sensible_heat_flux_vector);

  std::cout << " ==== latent_heat_flux" << std::endl;
  printVector2x2(products.latent_heat_flux_vector);

  std::cout << " ==== net_radiation_24h" << std::endl;
  printVector2x2(products.net_radiation_24h_vector);

  std::cout << " ==== evapotranspiration_fraction" << std::endl;
  printVector2x2(products.evapotranspiration_fraction_vector);

  std::cout << " ==== sensible_heat_flux_24h" << std::endl;
  printVector2x2(products.sensible_heat_flux_24h_vector);

  std::cout << " ==== latent_heat_flux_24h" << std::endl;
  printVector2x2(products.latent_heat_flux_24h_vector);

  std::cout << " ==== evapotranspiration_24h" << std::endl;
  printVector2x2(products.evapotranspiration_24h_vector);

  std::cout << " ==== evapotranspiration" << std::endl;
  printVector2x2(products.evapotranspiration_vector);

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "P3 - WRITE PRODUCTS," << general_time << "," << initial_time << "," << final_time << std::endl;
};

void Landsat::close()
{
  for (int i = 1; i <= 8; i++)
  {
    TIFFClose(this->bands_resampled[i]);
  }
};